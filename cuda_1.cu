#include "hip/hip_runtime.h"
#include<iostream>
using namespace std;
__global__ void MatrixMulKernel(float *Md,float *Nd,float *Pd,int Width){
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    float Pvalue=0;
    for(int k=0;k<Width;k++){
        float Mdelement=Md[ty*Width+k];
        float Ndelement=Nd[k*Width+ty];
        Pvalue+=Mdelement*Ndelement;
        
    }
    Pd[ty*Width+tx]=Pvalue;
}

void MatrixMultiplication(float *M,float *N,float *P,int Width){
     int size=Width*Width*sizeof(float);
     float* Md,*Nd,*Pd;
     hipMalloc((void **)&Md,size);
     hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
     hipMalloc((void **)&Nd,size);
     hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
     hipMalloc((void **)&Pd,size);
     dim3 dimBlock(Width,Width);
     dim3 dimGrid(1,1);
     MatrixMulKernel<<dimGrid,dimBlock>>(*Md,*Nd,*Pd,Width);
     hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);
     hipFree(Md);
     hipFree(Nd);
     hipFree(Pd);

 }
 int main(){
     float M[3][3]={1,2,3,4,5,6,7,8,9};
     float N[3][3]={9,8,7,6,5,4,3,2,1};
     float P[3][3]={0};
     MatrixMultiplication(*M,*N,*P,3);
     cout << "P[3][3] = " << endl;
     for(int m=0;m<3;m++){
         for(int n=0;n<3;n++){
             cout << P[m][n] << " ";ßß
         }
         cout << endl;
     }
 }