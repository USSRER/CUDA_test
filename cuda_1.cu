#include<iostream>
using namespace std;
 void MatrixMultiplication(float *M,float * N,float *P,int Width){
     int size=Width*Width*sizeof(float);
     float *Md,Nd,Pd;
     hipMalloc((void **)&Md,size);
     hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
     hipMalloc((void **)&Nd,size);
     hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
     hipMalloc((void **)&Pd,size)



    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);
    cudafree(Md);
    cudafree(Nd);
    cudafree(Pd);

 }